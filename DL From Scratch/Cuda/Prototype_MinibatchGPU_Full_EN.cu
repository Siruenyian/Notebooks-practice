// ============ MNIST Classifier from Scratch using GPU Acceleration ============
// This CUDA (.cu) file demonstrates the use of GPU acceleration for training a neural network.
// It applies a simple classifier to the MNIST dataset, leveraging CUDA kernels to perform
// key operations such as matrix multiplication, vector addition, and matrix transpotition efficiently on the GPU.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Loop to echeck Cuda availability
#define CUDA_CHECK(x)                                                                        \
    do                                                                                       \
    {                                                                                        \
        hipError_t err = x;                                                                 \
        if (err != hipSuccess)                                                              \
        {                                                                                    \
            printf("CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(1);                                                                         \
        }                                                                                    \
    } while (0)

// Fucntion to print Cuda Array on device
void CopyAndPrintDeviceArray(const float *device_array, int batchSize, int numClasses, const char *label)
{
    int total = batchSize * numClasses;
    float *host_array = (float *)malloc(total * sizeof(float));
    if (!host_array)
    {
        fprintf(stderr, "Failed to allocate host memory for %s\n", label);
        return;
    }

    hipError_t err = hipMemcpy(host_array, device_array, total * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA memcpy failed for %s: %s\n", label, hipGetErrorString(err));
        free(host_array);
        return;
    }

    printf("=== %s (batchSize = %d, numClasses = %d) ===\n", label, batchSize, numClasses);
    for (int b = 0; b < batchSize; ++b)
    {
        printf("Batch %d: ", b);
        for (int c = 0; c < numClasses; ++c)
        {
            printf("%.4f ", host_array[b * numClasses + c]);
        }
        printf("\n");
    }

    free(host_array);
}

// Mnist Image Loader Function
// Converts byte format to the correct configuration then read image file content
unsigned char *LoadMNISTImages(const char *path, int *number_of_images)
{
    FILE *f = fopen(path, "rb");
    if (!f)
    {
        perror("Could not open image file");
        exit(1);
    }

    int magic_number = 0;
    fread(&magic_number, sizeof(int), 1, f);
    // Fix and convert big endian to little for x86
    magic_number = __builtin_bswap32(magic_number);
    if (magic_number != 2051)
    {
        printf("Invalid magic number in image file: %d\n", magic_number);
        exit(1);
    }

    int numImages = 0, rows = 0, cols = 0;
    fread(&numImages, sizeof(int), 1, f);
    fread(&rows, sizeof(int), 1, f);
    fread(&cols, sizeof(int), 1, f);
    numImages = __builtin_bswap32(numImages);
    rows = __builtin_bswap32(rows);
    cols = __builtin_bswap32(cols);

    printf("Loading %d images of size %dx%d\n", numImages, rows, cols);

    unsigned char *images = (unsigned char *)malloc(numImages * rows * cols);
    fread(images, sizeof(unsigned char), numImages * rows * cols, f);
    fclose(f);

    *number_of_images = numImages;
    return images;
}

// Mnist Label Loader Function
// Converts byte format to the correct configuration then read label file content
unsigned char *LoadMnistLabels(const char *path, int *number_of_labels)
{
    FILE *f = fopen(path, "rb");
    if (!f)
    {
        perror("Could not open label file");
        exit(1);
    }
    int magic_number = 0;
    fread(&magic_number, sizeof(int), 1, f);
    // Fix and convert big endian to little for x86
    magic_number = __builtin_bswap32(magic_number);
    if (magic_number != 2049)
    {
        printf("Invalid magic number in label file: %d\n", magic_number);
        exit(1);
    }

    int numLabels = 0;
    fread(&numLabels, sizeof(int), 1, f);
    numLabels = __builtin_bswap32(numLabels);

    unsigned char *labels = (unsigned char *)malloc(numLabels);
    fread(labels, sizeof(unsigned char), numLabels, f);
    fclose(f);

    *number_of_labels = numLabels;
    return labels;
}

// Normalizes array value to 0.0 - 1.0
void Normalize(float *normalizedImg, unsigned char *images, int len)
{
    for (int i = 0; i < len; i++)
    {
        normalizedImg[i] = images[i] / 255.0f;
    }
}

// Encodes a class label to one-hot vector
void OneHotEncode(float *onehotLabels, unsigned char *labels, int len, int numClasses)
{
    for (int i = 0; i < len; i++)
    {
        for (int j = 0; j < numClasses; j++)
        {
            onehotLabels[i * numClasses + j] = (labels[i] == j) ? 1.0f : 0.0f;
        }
    }
}

// ReLU function
float ReLU(float x)
{
    return x > 0 ? x : 0;
}

// Derivative of ReLU function
float D_ReLu(float x)
{
    return x >= 0 ? 1 : 0;
}

// Sigmoid Function
float Sigmoid(float x)
{
    return 1.0f / (1.0f + expf(-x));
}

// TODO: Implement quick switching of activation fucntion
// Types to quickly switch between activation fucntions
typedef enum
{
    RELU,
    SIGMOID,
    NONE
} ActivationType;

// Struct for one layer
struct Layer
{
    int inFeature;
    int outFeature;
    float *Weight;
    float *Bias;
    float *X;
    float *z;
    ActivationType activation;
};

// Weight initialize function
// It sets the weight array to rand/max * sqroot(size) - (sqroot(size)/2)
void InitWeights(Layer *layer)
{
    int in = layer->inFeature;
    int out = layer->outFeature;
    int totalSize = in * out;

    float *h_weights = (float *)malloc(totalSize * sizeof(float));
    float scale = sqrtf(2.0f / totalSize);
    for (int i = 0; i < totalSize; i++)
    {
        h_weights[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
    CUDA_CHECK(hipMalloc(&layer->Weight, totalSize * sizeof(float)));
    CUDA_CHECK(hipMemcpy(layer->Weight, h_weights, totalSize * sizeof(float), hipMemcpyHostToDevice));
    free(h_weights);
}

// Bias initialize function
// Sets the bias array to zeros
void InitBias(struct Layer *layer)
{
    int out = layer->outFeature;

    float *h_bias = (float *)calloc(out, sizeof(float)); // initialized to zero already

    CUDA_CHECK(hipMalloc(&layer->Bias, out * sizeof(float)));

    CUDA_CHECK(hipMemcpy(layer->Bias, h_bias, out * sizeof(float), hipMemcpyHostToDevice));

    free(h_bias);
}

// Categorical Cross-Entropy is a loss function used for multi-class classification tasks.
// It measures the difference between the predicted probability distribution (from softmax) and the actual one-hot encoded label.
// The formula is: L = -sum(y_true[i] * log(y_pred[i])) for each class i
// - y_true[i] is 1 for the correct class, 0 otherwise (one-hot vector)
// - y_pred[i] is the predicted probability for class i
// This loss penalizes wrong confident predictions more heavily and encourages the model to assign high probability to the correct class.
float CalculateCCELoss(float *yPred, float *yTrue, int batchSize, int numClasses)
{
    float lossAvg = 0;
    for (int i = 0; i < batchSize; i++)
    {
        float Loss = 0.0f;
        for (int j = 0; j < numClasses; j++)
        {
            int idx = i * numClasses + j;
            if (yTrue[idx] > 0.0f)
            {
                Loss += yTrue[idx] * logf(yPred[idx] + 1e-7f);
            }
        }
        Loss = -Loss;
        lossAvg += Loss;
    }
    // this affects the output gradient
    lossAvg = lossAvg / batchSize;
    return lossAvg;
}

// ==========CUDA Accelerated Functions and Kernels==========

// Kernel to intialize random array value
__global__ void __CU_init_random(float *data, int size, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        data[idx] = hiprand_uniform(&state) * 2.0f - 1.0f;
    }
}

// Kernel to calculate the ReLU function
__global__ void __CU_ReLU(float *data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        data[idx] = fmaxf(data[idx], 0.0f);
    }
}

// Kernel to calculate the derivative of the ReLU function
__global__ void __CU_relu_derivative(float *grad, float *x, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        grad[idx] *= (x[idx] > 0) ? 1.0f : 0.0f;
    }
}

// Kernel for Matrix Multiplication
// A: [M x K] - input matrix 1
// B: [K x N] - input matrix 2
// C: [M x N] - output matrix
__global__ void __CU_MatmulBatch(float *C, float *A, float *B, int M, int K, int N)
{
    // Declare row and col so the kernel knows where to target
    // can be read as row skip value + target col index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // safeguard so it doesnt exceed the thread count
    if (row < M && col < N)
    {
        // Matmul addition part
        float val = 0.0f;
        for (int k = 0; k < K; ++k)
        {
            val += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = val;
    }
}

// Kernel for Matrix Transpose
// A: [M x K] - input matrix
// B: [K x M] - output matrix
__global__ void __CU_Transpose(float *B, const float *A, int M, int K)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // safeguard so it doesnt exceed the thread
    if (row < M && col < K)
    {
        B[col * M + row] = A[row * K + col];
    }
}

// Kernel fot Matrix Addition
// A: [M x N] - input matrix 1
// B: [M X N] - input matrix 2
// C: [M x N] - output matrix
__global__ void __CU_VecAdd(float *C, float *A, float *B, int M, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // safeguard so it doesnt exceed the thread
    if (row < M && col < N)
    {
        C[row * N + col] = A[row * N + col] + B[col];
    }
}

// Unified Kernel for updating parameter
// This implementation uses Gradient Descent
__global__ void __CU_UpdateParameter(float *W, const float *dEdW, float learningRate, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        W[idx] -= learningRate * dEdW[idx];
    }
}

// Utility Kernel to compute squared sum
__global__ void __CU_ComputeSquaredSum(float *grad, float *partialSum, int size)
{
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < size)
    {
        temp += grad[tid] * grad[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Reduction in shared memory
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        partialSum[blockIdx.x] = cache[0];
}

// Utility Kernel to scale a value
__global__ void __CU_ReduceAndScale(float *partialSum, float *scale, int gridSize, float threshold)
{
    float normSq = 0.0f;
    for (int i = 0; i < gridSize; ++i)
    {
        normSq += partialSum[i];
    }

    float norm = sqrtf(normSq);
    if (norm > threshold)
    {
        *scale = threshold / norm;
    }
    else
    {
        *scale = 1.0f; // no scaling needed
    }
}

// Kernel to scale a gradient by some scalar value
__global__ void __CU_ScaleGradient(float *grad, int size, float *scale)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        grad[idx] *= *scale;
    }
}

// Kernel to clip gradient value and keep it in a certain maximum range
void __CU_ClipGradientValue(float *grad, int size, float threshold)
{
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    float *d_partialSum, *d_scale;
    CUDA_CHECK(hipMalloc(&d_partialSum, gridSize * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_scale, sizeof(float)));

    // compute squared sum in blocks->reduce->scale
    __CU_ComputeSquaredSum<<<gridSize, blockSize>>>(grad, d_partialSum, size);
    __CU_ReduceAndScale<<<1, 1>>>(d_partialSum, d_scale, gridSize, threshold);
    __CU_ScaleGradient<<<gridSize, blockSize>>>(grad, size, d_scale);

    hipFree(d_partialSum);
    hipFree(d_scale);
}

// Kernel to compute the passing gradient on each layer(dEdz)
__global__ void __CU_Compute_dEdz(float *dEdz, const float *current_dEdy, const float *z, int size, bool apply_relu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        float dydx = apply_relu ? (z[i] > 0.0f ? 1.0f : 0.0f) : 1.0f;
        dEdz[i] = current_dEdy[i] * dydx;
    }
}

// Kernel to compute the bias gradient value
__global__ void __CU_ComputeBiasGradient(float *dEdb, const float *dEdz, int batchSize, int outF)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < outF)
    {
        float sum = 0.0f;
        for (int b = 0; b < batchSize; b++)
        {
            sum += dEdz[b * outF + j];
        }
        dEdb[j] = sum;
    }
}

// Forward Propagation function
// logits: raw softmax predition
// layerSequence: struct of layers
// numLayers: the number of layers present in the struct
// input: truth data of the batch
// batchSize: size of the batch that is processed
void ForwardProp(float *logits, Layer *layerSequence, int numLayers, float *input, int batchSize)
{
    // Find max feature size for buffer allocation
    int maxFeature = 0;
    for (int i = 0; i < numLayers; i++)
    {
        if (layerSequence[i].inFeature > maxFeature)
            maxFeature = layerSequence[i].inFeature;
        if (layerSequence[i].outFeature > maxFeature)
            maxFeature = layerSequence[i].outFeature;
    }

    float *temp1;
    size_t bufferSize = batchSize * maxFeature * sizeof(float);
    CUDA_CHECK(hipMalloc(&temp1, bufferSize));

    float *current = input;
    float *next = temp1;

    dim3 blockSize(16, 16);

    for (int layer = 0; layer < numLayers; layer++)
    {
        int inF = layerSequence[layer].inFeature;
        int outF = layerSequence[layer].outFeature;
        float *W = layerSequence[layer].Weight;
        float *b = layerSequence[layer].Bias;

        dim3 gridSizeMatmul((outF + blockSize.x - 1) / blockSize.x,
                            (batchSize + blockSize.y - 1) / blockSize.y);
        __CU_MatmulBatch<<<gridSizeMatmul, blockSize>>>(next, current, W, batchSize, inF, outF);

        dim3 gridSizeVecAdd((outF + blockSize.x - 1) / blockSize.x,
                            (batchSize + blockSize.y - 1) / blockSize.y);
        __CU_VecAdd<<<gridSizeVecAdd, blockSize>>>(next, next, b, batchSize, outF);

        // Store results for backprop (allocate once outside if needed)
        if (!layerSequence[layer].z) // allocate once if not already
            CUDA_CHECK(hipMalloc(&layerSequence[layer].z, batchSize * outF * sizeof(float)));
        if (!layerSequence[layer].X)
            CUDA_CHECK(hipMalloc(&layerSequence[layer].X, batchSize * inF * sizeof(float)));

        CUDA_CHECK(hipMemcpy(layerSequence[layer].z, next, batchSize * outF * sizeof(float), hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy(layerSequence[layer].X, current, batchSize * inF * sizeof(float), hipMemcpyDeviceToDevice));

        if (layer < numLayers - 1)
        {
            int size = batchSize * outF;
            int threadsPerBlock = 256;
            int blocksReLU = (size + threadsPerBlock - 1) / threadsPerBlock;
            __CU_ReLU<<<blocksReLU, threadsPerBlock>>>(next, size);
        }

        // Swap buffers for next layer input
        float *tmp = current;
        current = next;
        next = tmp;
    }

    int outputSize = batchSize * layerSequence[numLayers - 1].outFeature;
    hipMemcpy(logits, current, outputSize * sizeof(float), hipMemcpyDeviceToDevice);

    hipFree(temp1);
}

// Forward Propagation function
// layerSequence: struct of layers
// dEdy: The gradiet of the prediction and truth
// numLayers: the number of layers present in the struct
// batchSize: size of the batch that is processed
// learningRate: value of the update step
void BackwardProp(struct Layer *layerSequence, float *dEdy, int numLayers, int batchSize, float learningRate)
{
    // this is fine
    float *current_dEdy = dEdy;
    // CopyAndPrintDeviceArray(current_dEdy, batchSize, 10, "awiskiwiwi");
    for (int layer = numLayers - 1; layer >= 0; layer--)
    {
        int inF = layerSequence[layer].inFeature;
        int outF = layerSequence[layer].outFeature;

        float *X = layerSequence[layer].X;
        //  z empty
        float *z = layerSequence[layer].z;
        // CopyAndPrintDeviceArray(z, batchSize, 10, "awiskiwiwi");
        float *W = layerSequence[layer].Weight;
        float *b = layerSequence[layer].Bias;

        // Allocate GPU buffers
        float *d_XT, *d_dEdz, *d_dEdW, *d_dEdb, *d_WT, *d_new_dEdy;

        CUDA_CHECK(hipMalloc(&d_XT, batchSize * inF * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_dEdz, batchSize * outF * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_dEdW, inF * outF * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_dEdb, outF * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_WT, inF * outF * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_new_dEdy, batchSize * inF * sizeof(float)));

        // Compute dEdz = dEdy * dReLU/dz
        int totalOut = batchSize * outF;
        int blockSize = 256;
        int gridSize = (totalOut + blockSize - 1) / blockSize;

        __CU_Compute_dEdz<<<gridSize, blockSize>>>(d_dEdz, current_dEdy, z, totalOut, (layer < numLayers - 1));

        // // X^T
        dim3 dimBlock(16, 16);
        dim3 dimGrid((inF + 15) / 16, (batchSize + 15) / 16);

        __CU_Transpose<<<dimGrid, dimBlock>>>(d_XT, X, batchSize, inF);
        // // dEdW = X^T @ dEdz
        __CU_MatmulBatch<<<dim3((outF + 15) / 16, (inF + 15) / 16), dim3(16, 16)>>>(d_dEdW, d_XT, d_dEdz, inF, batchSize, outF);
        // // Clip gradients
        __CU_ClipGradientValue(d_dEdW, inF * outF, 0.5f);
        // // dEdb = mean of dEdz over batch
        // CopyAndPrintDeviceArray(d_dEdW, batchSize, 10, ("awiskiwiwi"));
        __CU_ComputeBiasGradient<<<(outF + 255) / 256, 256>>>(d_dEdb, d_dEdz, batchSize, outF);
        // // Update W and b
        __CU_UpdateParameter<<<(inF * outF + 255) / 256, 256>>>(W, d_dEdW, learningRate, inF * outF);
        __CU_UpdateParameter<<<(outF + 255) / 256, 256>>>(b, d_dEdb, learningRate, outF);

        // // W^T
        __CU_Transpose<<<dim3((inF + 15) / 16, (outF + 15) / 16), dim3(16, 16)>>>(d_WT, W, inF, outF);
        // CopyAndPrintDeviceArray(d_WT, 1, 10, ("awiskiwiwi"));

        // // new_dEdy = dEdz @ W^T
        __CU_MatmulBatch<<<dim3((inF + 15) / 16, (batchSize + 15) / 16), dim3(16, 16)>>>(d_new_dEdy, d_dEdz, d_WT, batchSize, outF, inF);

        // there is a zero somewhere here
        if (layer != numLayers - 1)
        {
            hipFree(current_dEdy);
        }

        current_dEdy = d_new_dEdy;

        // // Free temporary GPU buffers
        hipFree(d_XT);
        hipFree(d_dEdz);
        hipFree(d_dEdW);
        hipFree(d_dEdb);
        hipFree(d_WT);
        // // do NOT free d_new_dEdy here
    }
    hipFree(current_dEdy); // free after last layer
}

// Softmax Kernel implementation of the softmax Function
// raw values->probability distribution
__global__ void __CU_Softmax(float *input, float *output, int batch_size, int size)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch_size)
    {
        int offset = b * size;
        float max_val = input[offset];
        for (int i = 1; i < size; ++i)
        {
            max_val = fmaxf(max_val, input[offset + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < size; ++i)
        {
            output[offset + i] = expf(input[offset + i] - max_val);
            sum += output[offset + i];
        }

        for (int i = 0; i < size; ++i)
        {
            output[offset + i] = fmaxf(output[offset + i] / sum, 1e-7f); // clamp to avoid log(0)
        }
    }
}

void CU_SoftMaxBatch(float *device_input, float *device_output, int batchSize, int numClasses)
{
    int threads = 128;
    int blocks = (batchSize + threads - 1) / threads;

    __CU_Softmax<<<blocks, threads>>>(device_input, device_output, batchSize, numClasses);
    // CUDA_CHECK(cudaGetLastError()); // cillegal memeroy access here
    // CUDA_CHECK(cudaDeviceSynchronize());
}

// Kernel to calcualte the gradient of the crossentropy loss fucntion
__global__ void __CU_CrossEntropyGradient(float *dEdy, const float *logits, const float *y_batch, int total, int batchSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total)
    {
        dEdy[idx] = (logits[idx] - y_batch[idx]) / batchSize;
    }
}

// Kernel wrapper function to calculate the crossentropy gradient
void CU_CrossEntropyGradient(float *dEdy, const float *d_logits, const float *d_y_batch, int batchSize, int numClasses)
{
    int total = batchSize * numClasses;
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    __CU_CrossEntropyGradient<<<gridSize, blockSize>>>(dEdy, d_logits, d_y_batch, total, batchSize);
}

// Function to predict a label given an mnist(28x28) image
void PredictSingle(int inputIndex, struct Layer *layerSequence, int numLayers, float *X, unsigned char *label, int numClasses)
{
    int inputSize = 784;

    float *deviceInput = &X[inputIndex * inputSize];
    float *logitsSingle = (float *)calloc(numClasses, sizeof(float));
    float *singleInput = (float *)calloc(inputSize, sizeof(float));
    float *logitsRaw;
    float *logitsSoftmax;
    CUDA_CHECK(hipMalloc(&logitsRaw, sizeof(float) * numClasses));
    CUDA_CHECK(hipMalloc(&logitsSoftmax, sizeof(float) * numClasses));
    ForwardProp(logitsRaw, layerSequence, numLayers, deviceInput, 1);
    CU_SoftMaxBatch(logitsRaw, logitsSoftmax, 1, layerSequence[numLayers - 1].outFeature);
    CUDA_CHECK(hipMemcpy(logitsSingle, logitsSoftmax, sizeof(float) * numClasses, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(singleInput, deviceInput, sizeof(float) * inputSize, hipMemcpyDeviceToHost));

    printf("=======================\n");
    printf("Predicted probabilities:\n");
    int predictedClass = 0;
    float maxProb = logitsSingle[0];
    for (int i = 1; i < numClasses; i++)
    {
        if (logitsSingle[i] > maxProb)
        {
            maxProb = logitsSingle[i];
            predictedClass = i;
        }
    }

    printf("Predicted class: %d\n", predictedClass);
    printf("True label: %d\n", label[inputIndex]);
    for (int i = 0; i < 28 * 28; i++)
    {
        if (singleInput[i] > 0)
        {
            if (singleInput[i] > 0.5)
            {
                printf("# ");
            }
            else
            {
                printf("- ");
            }
        }
        else
        {
            printf("  ");
        }
        if ((i + 1) % 28 == 0)
            printf("\n");
    }
    free(logitsSingle);
    free(singleInput);
    CUDA_CHECK(hipFree(logitsRaw));
    CUDA_CHECK(hipFree(logitsSoftmax));
}

// Main program
int main(int argc, char const *argv[])
{
    srand(time(NULL));

    // Data Source: https://github.com/cvdfoundation/mnist?tab=readme-ov-file
    const char *imagePath = "/app/dataset/MNIST/train-images-idx3-ubyte";
    const char *labelPath = "/app/dataset/MNIST/train-labels-idx1-ubyte";

    int numImages = 0;
    int numLabels = 0;
    unsigned char *images = LoadMNISTImages(imagePath, &numImages);
    unsigned char *labels = LoadMnistLabels(labelPath, &numLabels);

    if (numImages != numLabels)
    {
        printf("Mismatch between images and labels count.\n");
        exit(1);
    }

    float *normalizedImg = (float *)calloc(28 * 28 * numImages, sizeof(float));
    float *onehotLabels = (float *)calloc(10 * numLabels, sizeof(float));
    // Normalize images to a 0-1 range
    Normalize(normalizedImg, images, 28 * 28 * numImages);
    // Encode labels to one ot vecotrs
    OneHotEncode(onehotLabels, labels, numLabels, 10);

    float *X, *y;
    CUDA_CHECK(hipMalloc(&X, sizeof(float) * 28 * 28 * numImages));
    CUDA_CHECK(hipMemcpy(X, normalizedImg, sizeof(float) * 28 * 28 * numImages, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&y, sizeof(float) * 10 * numLabels));
    CUDA_CHECK(hipMemcpy(y, onehotLabels, sizeof(float) * 10 * numLabels, hipMemcpyHostToDevice));
    // Print Example Image
    printf("Example image\n");
    printf("Label[0] = %d\n", labels[0]);
    for (int i = 0; i < 28 * 28; i++)
    {
        if (normalizedImg[i] > 0)
        {
            printf("# ");
        }
        else
        {
            printf("- ");
        }
        if ((i + 1) % 28 == 0)
            printf("\n");
    }

    // Build Layers
    struct Layer layerSequence[] = {
        {.inFeature = 784, .outFeature = 4096, .activation = RELU},
        {.inFeature = 4096, .outFeature = 10, .activation = NONE}};
    int numLayers = sizeof(layerSequence) / sizeof(layerSequence[0]);

    printf("Log: There are %d layers in this network\n", numLayers);
    // Intialize weight and bias values
    for (int i = 0; i < numLayers; i++)
    {
        InitWeights(&layerSequence[i]);
        InitBias(&layerSequence[i]);
    }
    int numClasses = 10;
    int totalSamples = 10000;
    int epoch_count = 20;
    int batchSize = 32;
    int batchCount = totalSamples / batchSize;

    // Allocate buffers for logits and output gradient
    float *logits;
    float *logits_softmax;
    float *dEdy;
    CUDA_CHECK(hipMalloc(&logits, sizeof(float) * batchSize * numClasses));
    CUDA_CHECK(hipMalloc(&logits_softmax, sizeof(float) * batchSize * numClasses));
    CUDA_CHECK(hipMalloc(&dEdy, sizeof(float) * batchSize * numClasses));

    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    float *logits_host = (float *)malloc(batchSize * numClasses * sizeof(float));
    float *y_batch_host = (float *)malloc(batchSize * numClasses * sizeof(float));
    for (int epoch = 0; epoch < epoch_count; epoch++)
    {
        double totalLoss = 0.0;

        for (int b = 0; b < batchCount; b++)
        {
            float *X_batch = &X[b * batchSize * 784];
            float *y_batch = &y[b * batchSize * numClasses];
            ForwardProp(logits, layerSequence, numLayers, X_batch, batchSize);
            CU_SoftMaxBatch(logits, logits_softmax, batchSize, numClasses);
            CU_CrossEntropyGradient(dEdy, logits_softmax, y_batch, batchSize, numClasses);
            CUDA_CHECK(hipMemcpy(logits_host, logits_softmax, batchSize * numClasses * sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(y_batch_host, y_batch, batchSize * numClasses * sizeof(float), hipMemcpyDeviceToHost));
            totalLoss += CalculateCCELoss(logits_host, y_batch_host, batchSize, numClasses);
            BackwardProp(layerSequence, dEdy, numLayers, batchSize, 0.05f);
        }

        printf("Epoch #%d, Avg Loss: %.6f\n", epoch + 1, totalLoss / batchCount);
    }
    free(logits_host);
    free(y_batch_host);

    clock_gettime(CLOCK_MONOTONIC, &end);

    // Calculate duration in seconds with milliseconds
    double training_time = (end.tv_sec - start.tv_sec) +
                           (end.tv_nsec - start.tv_nsec) / 1e9;
    printf("\nTotal training time: %.2f sec\n", training_time);

    // Print predictions and check learning result
    PredictSingle(1250, layerSequence, numLayers, X, labels, numClasses);
    PredictSingle(49230, layerSequence, numLayers, X, labels, numClasses);
    PredictSingle(25122, layerSequence, numLayers, X, labels, numClasses);

    // Free memory - this throws illegal for some reason????
    CUDA_CHECK(hipFree(logits));
    CUDA_CHECK(hipFree(logits_softmax));
    CUDA_CHECK(hipFree(dEdy));

    free(images);
    free(labels);
    free(normalizedImg);
    free(onehotLabels);

    printf("=====program end=====\n");
    return 0;
}