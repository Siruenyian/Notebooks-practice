// ============ MNIST Classifier from Scratch using GPU Acceleration ============
// This CUDA (.cu) file demonstrates the use of GPU acceleration for training a neural network.
// It applies a simple classifier to the MNIST dataset, leveraging CUDA kernels to perform
// key operations such as matrix multiplication, vector addition, and matrix transpotition efficiently on the GPU.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
// Global Variable to set GPU Acceleration
// Initialized as False, do not change this
// Change the one on the main fcuntion instead
bool gpuInference = false;

// Mnist Image Loader Function
// Converts byte format to the correct configuration then read image file content
unsigned char *LoadMNISTImages(const char *path, int *number_of_images)
{
    FILE *f = fopen(path, "rb");
    if (!f)
    {
        perror("Could not open image file");
        exit(1);
    }

    int magic_number = 0;
    fread(&magic_number, sizeof(int), 1, f);
    // Fix and convert big endian to little for x86
    magic_number = __builtin_bswap32(magic_number);
    if (magic_number != 2051)
    {
        printf("Invalid magic number in image file: %d\n", magic_number);
        exit(1);
    }

    int numImages = 0, rows = 0, cols = 0;
    fread(&numImages, sizeof(int), 1, f);
    fread(&rows, sizeof(int), 1, f);
    fread(&cols, sizeof(int), 1, f);
    numImages = __builtin_bswap32(numImages);
    rows = __builtin_bswap32(rows);
    cols = __builtin_bswap32(cols);

    printf("Loading %d images of size %dx%d\n", numImages, rows, cols);

    unsigned char *images = (unsigned char *)malloc(numImages * rows * cols);
    fread(images, sizeof(unsigned char), numImages * rows * cols, f);
    fclose(f);

    *number_of_images = numImages;
    return images;
}

// Mnist Label Loader Function
// Converts byte format to the correct configuration then read label file content
unsigned char *LoadMnistLabels(const char *path, int *number_of_labels)
{
    FILE *f = fopen(path, "rb");
    if (!f)
    {
        perror("Could not open label file");
        exit(1);
    }
    int magic_number = 0;
    fread(&magic_number, sizeof(int), 1, f);
    // Fix and convert big endian to little for x86
    magic_number = __builtin_bswap32(magic_number);
    if (magic_number != 2049)
    {
        printf("Invalid magic number in label file: %d\n", magic_number);
        exit(1);
    }

    int numLabels = 0;
    fread(&numLabels, sizeof(int), 1, f);
    numLabels = __builtin_bswap32(numLabels);

    unsigned char *labels = (unsigned char *)malloc(numLabels);
    fread(labels, sizeof(unsigned char), numLabels, f);
    fclose(f);

    *number_of_labels = numLabels;
    return labels;
}

// ReLU function
float ReLU(float x)
{
    return x > 0 ? x : 0;
}

// Derivative of ReLU function
float D_ReLu(float x)
{
    return x >= 0 ? 1 : 0;
}

// Sigmoid Function
float Sigmoid(float x)
{
    return 1.0f / (1.0f + expf(-x));
}

// Softmax Function
// modified to accept batches
void SoftMaxBatch(float *x, int batchSize, int size)
{
    for (int b = 0; b < batchSize; ++b)
    {
        float *row = &x[b * size];

        // Step 1: find max for numerical stability
        float max_val = row[0];
        for (int i = 1; i < size; i++)
        {
            if (row[i] > max_val)
                max_val = row[i];
        }

        // Step 2: exponentiate and sum
        float sum = 0.0f;
        for (int i = 0; i < size; i++)
        {
            row[i] = expf(row[i] - max_val);
            sum += row[i];
        }

        // Step 3: normalize
        for (int i = 0; i < size; i++)
        {
            row[i] /= sum;
        }
    }
}

// TODO: Implement quick switching of activation fucntion
// Types to quickly switch between activation fucntions
typedef enum
{
    RELU,
    SIGMOID,
    NONE
} ActivationType;

// Struct for one layer
struct Layer
{
    int inFeature;
    int outFeature;
    float *Weight;
    float *Bias;
    float *X;
    float *z;
    ActivationType activation;
};

// Returns a random floating point(decimal) value
float randFloat()
{
    return ((float)rand()) / RAND_MAX;
}

// A: [M x K] - input matrix 1
// B: [K x N] - input matrix 2
// C: [M x N] - output
void MatmulBatch(float *C, float *A, float *B, int M, int K, int N)
{
    for (int m = 0; m < M; ++m)
    {
        float *a = &A[m * K];
        float *c = &C[m * N];

        for (int n = 0; n < N; ++n)
        {
            float val = 0.0f;
            for (int k = 0; k < K; ++k)
            {
                val += B[k * N + n] * a[k];
            }
            c[n] = val;
        }
    }
}

// A: [M x K] - input matrix
// B: [K x M] - output matrix
void Transpose(float *B, const float *A, int rows, int cols)
{
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            // B[j][i] = A[i][j]
            B[j * rows + i] = A[i * cols + j];
        }
    }
}

void InitWeights(struct Layer *layer)
{
    int in = layer->inFeature;
    int out = layer->outFeature;

    // Allocate a 1D array of size (inFeature * outFeature)
    // Weight is shaped (inFeature, outFeature)
    layer->Weight = (float *)malloc(in * out * sizeof(float));
    int totalSize = in * out;
    float scale = sqrtf(2.0f / totalSize);
    for (int i = 0; i < totalSize; i++)
    {
        layer->Weight[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
}

void InitBias(struct Layer *layer)
{
    int out = layer->outFeature;

    // Allocate a 1D array of size outFeature
    // Bias is shaped (outFeature)
    layer->Bias = (float *)calloc(out, sizeof(float));

    for (int i = 0; i < out; ++i)
    {
        layer->Bias[i] = 0;
    }
}

// Categorical Cross-Entropy is a loss function used for multi-class classification tasks.
// It measures the difference between the predicted probability distribution (from softmax) and the actual one-hot encoded label.
// The formula is: L = -sum(y_true[i] * log(y_pred[i])) for each class i
// - y_true[i] is 1 for the correct class, 0 otherwise (one-hot vector)
// - y_pred[i] is the predicted probability for class i
// This loss penalizes wrong confident predictions more heavily and encourages the model to assign high probability to the correct class.
float CalculateCCELoss(float *yPred, float *yTrue, int batchSize, int numClasses)
{
    float lossAvg = 0;
    for (int i = 0; i < batchSize; i++)
    {
        float Loss = 0.0f;
        for (int j = 0; j < numClasses; j++)
        {
            int idx = i * numClasses + j;
            if (yTrue[idx] > 0.0f)
            {
                Loss += yTrue[idx] * logf(yPred[idx] + 1e-7f);
            }
        }
        Loss = -Loss;
        lossAvg += Loss;
    }
    // this affects the output gradient
    lossAvg = lossAvg / batchSize;
    return lossAvg;
}

// ==========CUDA Accelerated Functions and Kernels==========

__global__ void CU_init_random(float *data, int size, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        data[idx] = hiprand_uniform(&state) * 2.0f - 1.0f;
    }
}

__global__ void CU_relu_derivative(float *grad, float *x, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        grad[idx] *= (x[idx] > 0) ? 1.0f : 0.0f;
    }
}
// A: [M x K] - input matrix 1
// B: [K x N] - input matrix 2
// C: [M x N] - output matrix
__global__ void __CU_MatmulBatch(float *C, float *A, float *B, int M, int K, int N)
{
    // Declare row and col so the kernel knows where to target
    // can be read as row skip value + target col index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // safeguard so it doesnt exceed the thread count
    if (row < M && col < N)
    {
        // Matmul addition part
        float val = 0.0f;
        for (int k = 0; k < K; ++k)
        {
            val += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = val;
    }
}

// Cuda Matmul wrapper function
void CU_MatmulBatch(float *C, float *A, float *B, int M, int K, int N)
{
    // Allocate Memory
    float *d_A;
    float *d_B;
    float *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));
    // Copy from CPU->GPU
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Declare block and grid dimensions(parameters)
    // Hierarchy: grid->block->thread
    // remember: cuda grid is written as (col, row) so its x first
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y);

    // Call cuda Kernel
    __CU_MatmulBatch<<<gridDim, blockDim>>>(d_C, d_A, d_B, M, K, N);

    // Wait Until all execution is finished
    hipDeviceSynchronize();

    // Copy from GPU->CPU
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// A: [M x K] - input matrix
// B: [K x M] - output matrix
__global__ void __CU_Transpose(float *B, const float *A, int M, int K)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // safeguard so it doesnt exceed the thread
    if (row < M && col < K)
    {
        B[col * M + row] = A[row * K + col];
    }
}

// Cuda Transpose wrapper function
void CU_Transpose(float *B, const float *A, int M, int K)
{
    float *d_A;
    // B is out
    float *d_B;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, M * K * sizeof(float));
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * K * sizeof(float), hipMemcpyHostToDevice);
    dim3 blockDim(16, 16);
    // remember: cuda grid is col, row, so its x first
    dim3 gridDim((K + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y);
    __CU_Transpose<<<gridDim, blockDim>>>(d_B, d_A, M, K);
    hipDeviceSynchronize();

    hipMemcpy(B, d_B, M * K * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
}

// A: [M x N] - input matrix 1
// B: [M X N] - input matrix 2
// C: [M x N] - output matrix
__global__ void __CU_VecAdd(float *C, float *A, float *B, int M, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // safeguard so it doesnt exceed the thread
    if (row < M && col < N)
    {
        C[row * N + col] = A[row * N + col] + B[col];
    }
}

void CU_VecAdd(float *C, float *A, float *B, int M, int N)
{
    float *d_A;
    float *d_B;
    float *d_C;

    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    // grid->block->thread
    // max
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y);

    __CU_VecAdd<<<gridDim, blockDim>>>(d_C, d_A, d_B, M, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void __CU_UpdateParameter(float *W, const float *dEdW, float learningRate, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        W[idx] -= learningRate * dEdW[idx];
    }
}

void CU_UpdateParameter(float *W, const float *dEdW, float learningRate, int M, int K)
{
    float *d_W, *d_dEdW;
    int size = M * K;
    hipMalloc(&d_W, size * sizeof(float));
    hipMalloc(&d_dEdW, size * sizeof(float));

    hipMemcpy(d_W, W, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dEdW, dEdW, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    // just one grid
    int gridSize = (size + blockSize - 1) / blockSize;

    __CU_UpdateParameter<<<gridSize, blockSize>>>(d_W, d_dEdW, learningRate, size);
    hipDeviceSynchronize();

    hipMemcpy(W, d_W, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_W);
    hipFree(d_dEdW);
}

__global__ void ComputeSquaredSum(const float *grad, float *partialSum, int size)
{
    __shared__ float sharedSum[256];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sharedSum[tid] = (i < size) ? grad[i] * grad[i] : 0.0f;

    __syncthreads();

    // Reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sharedSum[tid] += sharedSum[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        partialSum[blockIdx.x] = sharedSum[0];
    }
}

__global__ void ScaleGradient(float *grad, int size, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        grad[i] *= scale;
    }
}

void CU_ClipGradientValue(float *grad, int size, float threshold)
{
    float *d_grad, *d_partialSum;
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    hipMalloc(&d_grad, size * sizeof(float));
    hipMemcpy(d_grad, grad, size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_partialSum, gridSize * sizeof(float));
    ComputeSquaredSum<<<gridSize, blockSize>>>(d_grad, d_partialSum, size);
    hipDeviceSynchronize();

    float *h_partialSum = (float *)malloc(gridSize * sizeof(float));
    hipMemcpy(h_partialSum, d_partialSum, gridSize * sizeof(float), hipMemcpyDeviceToHost);

    float normSq = 0.0f;
    for (int i = 0; i < gridSize; i++)
    {
        normSq += h_partialSum[i];
    }
    float norm = sqrtf(normSq);

    if (norm > threshold)
    {
        float scale = threshold / norm;
        ScaleGradient<<<gridSize, blockSize>>>(d_grad, size, scale);
        hipDeviceSynchronize();
    }

    hipMemcpy(grad, d_grad, size * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_grad);
    hipFree(d_partialSum);
    free(h_partialSum);
}

__global__ void __CU_Compute_dEdz(float *dEdz, const float *current_dEdy, const float *z, int size, bool apply_relu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        float dydx = apply_relu ? (z[i] > 0.0f ? 1.0f : 0.0f) : 1.0f;
        dEdz[i] = current_dEdy[i] * dydx;
    }
}

void CU_Compute_dEdz(float *dEdz_host, const float *current_dEdy_host, const float *z_host, int batchSize, int outF, bool apply_relu)
{
    int size = batchSize * outF;
    size_t bytes = size * sizeof(float);

    // Allocate device memory
    float *d_dEdz, *d_current_dEdy, *d_z;
    hipMalloc(&d_dEdz, bytes);
    hipMalloc(&d_current_dEdy, bytes);
    hipMalloc(&d_z, bytes);

    // Copy host data to device
    hipMemcpy(d_current_dEdy, current_dEdy_host, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z_host, bytes, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    __CU_Compute_dEdz<<<gridSize, blockSize>>>(d_dEdz, d_current_dEdy, d_z, size, apply_relu);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(dEdz_host, d_dEdz, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_dEdz);
    hipFree(d_current_dEdy);
    hipFree(d_z);
}
__global__ void CU_MatmulAddBiasKernel(
    float *next,                       // [batchSize, outF]
    const float *__restrict__ current, // [batchSize, inF]
    const float *__restrict__ W,       // [inF, outF]
    const float *__restrict__ bias,    // [outF]
    int batchSize,
    int inF,
    int outF)
{
    int o = blockIdx.x * blockDim.x + threadIdx.x; // output feature index
    int b = blockIdx.y * blockDim.y + threadIdx.y; // batch index

    if (b < batchSize && o < outF)
    {
        float sum = 0.0f;
        for (int i = 0; i < inF; ++i)
        {
            sum += current[b * inF + i] * W[i * outF + o];
        }
        next[b * outF + o] = sum + bias[o];
    }
}

void ForwardProp(float *logits, struct Layer *layerSequence, int numLayers, float *input, int batchSize)
{

    // allocate 2 buffers
    float *temp1 = (float *)calloc(4096 * batchSize, sizeof(float));
    float *temp2 = (float *)calloc(4096 * batchSize, sizeof(float));
    float *current = input;
    float *next = temp1;
    for (int layer = 0; layer < numLayers; layer++)
    {
        int inF = layerSequence[layer].inFeature;
        int outF = layerSequence[layer].outFeature;
        float *W = layerSequence[layer].Weight;
        float *b = layerSequence[layer].Bias;

        // 32->the neuron count
        // 30->the connection on each neuron, same as length of a single training item

        // Multiply current @ W  -> next
        if (gpuInference)
        {
            CU_MatmulBatch(next, current, W, batchSize, inF, outF);
            CU_VecAdd(next, next, b, batchSize, outF);
        }
        else
        {
            MatmulBatch(next, current, W, batchSize, inF, outF);
            for (int i = 0; i < batchSize; i++)
            {
                for (int j = 0; j < outF; j++)
                {
                    next[i * outF + j] += b[j];
                }
            }
        }

        layerSequence[layer].z = (float *)calloc(batchSize * outF, sizeof(float));
        layerSequence[layer].X = (float *)calloc(batchSize * inF, sizeof(float));
        memcpy(layerSequence[layer].z, next, batchSize * outF * sizeof(float));
        memcpy(layerSequence[layer].X, current, batchSize * inF * sizeof(float));

        // Apply ReLU element-wise
        if (layer < numLayers - 1)
        {
            for (int i = 0; i < batchSize * outF; i++)
            {
                next[i] = ReLU(next[i]);
            }
        }

        float *temp = current;
        current = next;
        if (layer < numLayers - 1)
        {
            next = (temp == temp1) ? temp2 : temp1;
        }
    }

    // plain copy value
    memcpy(logits, next, batchSize * layerSequence[numLayers - 1].outFeature * sizeof(float));
    free(temp1);
    free(temp2);
}

void ClipGradientValue(float *grad, int size, float threshold)
{
    float norm = 0.0f;
    for (int i = 0; i < size; i++)
    {
        norm += grad[i] * grad[i];
    }
    norm = sqrtf(norm);

    if (norm > threshold)
    {
        float scale = threshold / norm;
        for (int i = 0; i < size; i++)
        {
            grad[i] *= scale;
        }
    }
}

__global__ void __CU_ComputeBiasGradient(float *dEdb, const float *dEdz, int batchSize, int outF)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < outF)
    {
        float sum = 0.0f;
        for (int b = 0; b < batchSize; b++)
        {
            sum += dEdz[b * outF + j];
        }
        dEdb[j] = sum;
    }
}

void CU_ComputeBiasGradient(float *dEdb_host, const float *dEdz_device, int batchSize, int outF)
{
    float *d_dEdb;
    hipMalloc(&d_dEdb, outF * sizeof(float));

    int blockSize = 256;
    int gridSize = (outF + blockSize - 1) / blockSize;
    __CU_ComputeBiasGradient<<<gridSize, blockSize>>>(d_dEdb, dEdz_device, batchSize, outF);
    hipDeviceSynchronize();

    hipMemcpy(dEdb_host, d_dEdb, outF * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_dEdb);
}
#define CUDA_CHECK(x)                                                                        \
    do                                                                                       \
    {                                                                                        \
        hipError_t err = x;                                                                 \
        if (err != hipSuccess)                                                              \
        {                                                                                    \
            printf("CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(1);                                                                         \
        }                                                                                    \
    } while (0)

void CU_BackpropagateLayer(
    float *W, float *b,
    const float *X, const float *z,
    const float *current_dEdy,
    float *new_dEdy,
    int batchSize, int inF, int outF,
    float *dEdW, float *dEdb,
    float learningRate, float clipValue,
    bool apply_relu)
{
    int totalOut = batchSize * outF;
    int totalIn = batchSize * inF;

    float *d_X, *d_XT, *d_z, *d_current_dEdy, *d_dEdz;
    float *d_W, *d_WT, *d_new_dEdy, *d_dEdW, *d_dEdb, *d_b;

    CUDA_CHECK(hipMalloc(&d_X, totalIn * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_XT, totalIn * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_z, totalOut * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_current_dEdy, totalOut * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dEdz, totalOut * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_W, inF * outF * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_WT, inF * outF * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_new_dEdy, batchSize * inF * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dEdW, inF * outF * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dEdb, outF * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, outF * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_X, X, totalIn * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_z, z, totalOut * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_current_dEdy, current_dEdy, totalOut * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_W, W, inF * outF * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, outF * sizeof(float), hipMemcpyHostToDevice));

    // Compute dEdz = current_dEdy * dReLU/dz (if relu)
    int blockSize = 256;
    int gridSize = (totalOut + blockSize - 1) / blockSize;
    __CU_Compute_dEdz<<<gridSize, blockSize>>>(d_dEdz, d_current_dEdy, d_z, totalOut, apply_relu);

    // Transpose X -> XT
    dim3 dimBlock(16, 16);
    dim3 dimGrid((inF + 15) / 16, (batchSize + 15) / 16);
    __CU_Transpose<<<dimGrid, dimBlock>>>(d_XT, d_X, batchSize, inF);

    // Compute dEdW = XT @ dEdz
    dim3 gridMatmul((outF + 15) / 16, (inF + 15) / 16);
    dim3 blockMatmul(16, 16);
    __CU_MatmulBatch<<<gridMatmul, blockMatmul>>>(d_dEdW, d_XT, d_dEdz, inF, batchSize, outF);

    // Clip gradients
    // Clip gradients
    CU_ClipGradientValue(d_dEdW, inF * outF, clipValue);

    // Compute dEdb = average over batch of dEdz
    __CU_ComputeBiasGradient<<<(outF + 255) / 256, 256>>>(d_dEdb, d_dEdz, batchSize, outF);

    // Update weights and bias
    __CU_UpdateParameter<<<(inF * outF + 255) / 256, 256>>>(d_W, d_dEdW, learningRate, inF * outF);
    __CU_UpdateParameter<<<(outF + 255) / 256, 256>>>(d_b, d_dEdb, learningRate, outF);

    // Transpose W -> WT
    __CU_Transpose<<<dim3((inF + 15) / 16, (outF + 15) / 16), dim3(16, 16)>>>(d_WT, d_W, inF, outF);

    // Compute new_dEdy = dEdz @ WT
    __CU_MatmulBatch<<<dim3((inF + 15) / 16, (batchSize + 15) / 16), dim3(16, 16)>>>(d_new_dEdy, d_dEdz, d_WT, batchSize, outF, inF);

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(W, d_W, inF * outF * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(b, d_b, outF * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(new_dEdy, d_new_dEdy, batchSize * inF * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(dEdW, d_dEdW, inF * outF * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(dEdb, d_dEdb, outF * sizeof(float), hipMemcpyDeviceToHost));

    // Cleanup
    hipFree(d_X);
    hipFree(d_XT);
    hipFree(d_z);
    hipFree(d_current_dEdy);
    hipFree(d_dEdz);
    hipFree(d_W);
    hipFree(d_WT);
    hipFree(d_new_dEdy);
    hipFree(d_dEdW);
    hipFree(d_dEdb);
    hipFree(d_b);
}

void BackwardProp(struct Layer *layerSequence, float *dEdy, int numLayers, int batchSize, float learningRate)
{
    float *current_dEdy = dEdy;

    for (int layer = numLayers - 1; layer >= 0; layer--)
    {
        int inF = layerSequence[layer].inFeature;
        int outF = layerSequence[layer].outFeature;
        float *W = layerSequence[layer].Weight;
        float *b = layerSequence[layer].Bias;
        float *X = layerSequence[layer].X;
        float *z = layerSequence[layer].z;
        float *dydx = (float *)calloc(batchSize * outF, sizeof(float));
        float *dEdz = (float *)calloc(batchSize * outF, sizeof(float));
        float *dEdW = (float *)calloc(inF * outF, sizeof(float));
        float *dEdb = (float *)calloc(outF, sizeof(float));
        // X is [inF x batchSize], XT is [batchSize x inF]
        float *XT = (float *)malloc(inF * batchSize * sizeof(float));
        // W is [inF x outF], WT is [outF x inF]
        float *WT = (float *)malloc(outF * inF * sizeof(float));
        float *new_dEdy = (float *)malloc(batchSize * inF * sizeof(float));
        // Clip value/the maximum of a gradient value can reach
        float clipValue = 0.5;
        if (gpuInference)
        {
            CU_BackpropagateLayer(
                W, b,
                X, z,
                current_dEdy,
                new_dEdy,
                batchSize, inF, outF,
                dEdW, dEdb,
                learningRate, clipValue,
                true);
        }
        else
        {

            for (int i = 0; i < batchSize * outF; i++)
            {
                dydx[i] = (layer < numLayers - 1) ? D_ReLu(z[i]) : 1.0f;
                dEdz[i] = current_dEdy[i] * dydx[i];
            }
            Transpose(XT, X, batchSize, inF);
            // dEdz @ self.W.T
            MatmulBatch(dEdW, XT, dEdz, inF, batchSize, outF);
            ClipGradientValue(dEdW, inF * outF, clipValue);
            // printf("layer %d: %.2lf, %.2lf, %.2lf\n", layer, X[0], W[0], z[0]);
            // printf("layer %d deriv: %.2lf, %.2lf, %.2lf\n", layer, dEdz[0], dEdW[0], current_dEdy[0]);
            for (int j = 0; j < outF; j++)
            {
                dEdb[j] = 0.0f;
                for (int b = 0; b < batchSize; b++)
                {
                    dEdb[j] += dEdz[b * outF + j];
                }
            }
            for (int i = 0; i < inF * outF; i++)
            {
                W[i] -= learningRate * dEdW[i];
            }
            for (int j = 0; j < outF; j++)
            {
                b[j] -= learningRate * dEdb[j];
            }
            Transpose(WT, W, inF, outF);
            MatmulBatch(new_dEdy, dEdz, WT, batchSize, outF, inF);
        }

        if (layer != numLayers - 1)
        {
            free(current_dEdy);
        }

        current_dEdy = new_dEdy;

        free(XT);
        free(WT);
        free(dydx);
        free(dEdz);
        free(dEdW);
        free(dEdb);
    }

    free(current_dEdy);
}

// Normalizes array value to 0.0 - 1.0
void Normalize(float *normalizedImg, unsigned char *images, int len)
{
    for (int i = 0; i < len; i++)
    {
        normalizedImg[i] = images[i] / 255.0f;
    }
}

// Encodes a class label to one-hot vector
void OneHotEncode(float *onehotLabels, unsigned char *labels, int len, int numClasses)
{
    for (int i = 0; i < len; i++)
    {
        for (int j = 0; j < numClasses; j++)
        {
            onehotLabels[i * numClasses + j] = (labels[i] == j) ? 1.0f : 0.0f;
        }
    }
}

void PredictSingle(int inputIndex, struct Layer *layerSequence, int numLayers, float *X, unsigned char *label, int numClasses)
{
    int inputSize = 784;

    float *singleInput = &X[inputIndex * inputSize];
    float *logitsSingle = (float *)calloc(numClasses, sizeof(float));

    ForwardProp(logitsSingle, layerSequence, numLayers, singleInput, 1);
    SoftMaxBatch(logitsSingle, 1, layerSequence[numLayers - 1].outFeature);
    printf("=======================\n");
    printf("Predicted probabilities:\n");
    int predictedClass = 0;
    float maxProb = logitsSingle[0];
    for (int i = 1; i < numClasses; i++)
    {
        if (logitsSingle[i] > maxProb)
        {
            maxProb = logitsSingle[i];
            predictedClass = i;
        }
    }

    printf("Predicted class: %d\n", predictedClass);
    printf("True label: %d\n", label[inputIndex]);
    for (int i = 0; i < 28 * 28; i++)
    {
        if (singleInput[i] > 0)
        {
            printf("# ");
        }
        else
        {
            printf("- ");
        }
        if ((i + 1) % 28 == 0)
            printf("\n");
    }
    free(logitsSingle);
}

__global__ void softmax_kernel(float *x, int batch_size, int size)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch_size)
    {
        float max_val = x[b * size];
        for (int i = 1; i < size; ++i)
        {
            max_val = fmaxf(max_val, x[b * size + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < size; ++i)
        {
            x[b * size + i] = expf(x[b * size + i] - max_val);
            sum += x[b * size + i];
        }

        for (int i = 0; i < size; ++i)
        {
            x[b * size + i] = fmaxf(x[b * size + i] / sum, 1e-7f); // clamp to avoid log(0)
        }
    }
}

void CU_SoftMaxBatch(float *host_logits, int batchSize, int numClasses)
{
    int total = batchSize * numClasses;
    float *device_logits;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&device_logits, total * sizeof(float)));

    // Copy input logits from host to device
    CUDA_CHECK(hipMemcpy(device_logits, host_logits, total * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    int threads = 128;
    int blocks = (batchSize + threads - 1) / threads;
    softmax_kernel<<<blocks, threads>>>(device_logits, batchSize, numClasses);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(host_logits, device_logits, total * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(device_logits));
}

__global__ void __CU_CrossEntropyGradient(float *dEdy, const float *logits, const float *y_batch, int total, int batchSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total)
    {
        dEdy[idx] = (logits[idx] - y_batch[idx]) / batchSize;
    }
}

void CU_CrossEntropyGradient(float *dEdy_host, const float *logits_host, const float *y_batch_host, int batchSize, int numClasses)
{
    int total = batchSize * numClasses;
    float *d_logits, *d_y_batch, *d_dEdy;

    // Allocate memory
    CUDA_CHECK(hipMalloc(&d_logits, total * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y_batch, total * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dEdy, total * sizeof(float)));

    // Copy inputs to device
    CUDA_CHECK(hipMemcpy(d_logits, logits_host, total * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y_batch, y_batch_host, total * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    __CU_CrossEntropyGradient<<<gridSize, blockSize>>>(d_dEdy, d_logits, d_y_batch, total, batchSize);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    CUDA_CHECK(hipMemcpy(dEdy_host, d_dEdy, total * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    hipFree(d_logits);
    hipFree(d_y_batch);
    hipFree(d_dEdy);
}

int main(int argc, char const *argv[])
{
    // Toggle this for GPU Acceleration

    if (argc == 2)
    {
        printf("The argument supplied is %s\n", argv[1]);
        if (strcmp("true", argv[1]) == 0)
        {

            gpuInference = true;
            printf("Using GPU Inference\n");
        }
        else
        {
            printf("Using CPU Inference\n");
        }
    }
    else if (argc > 2)
    {
        printf("Too many arguments supplied.\n");
        return 0;
    }
    else
    {
        printf("One argument expected.\n");
        return 0;
    }

    // Data Source: https://github.com/cvdfoundation/mnist?tab=readme-ov-file
    const char *imagePath = "/app/dataset/MNIST/train-images-idx3-ubyte";
    const char *labelPath = "/app/dataset/MNIST/train-labels-idx1-ubyte";

    int numImages = 0;
    int numLabels = 0;
    unsigned char *images = LoadMNISTImages(imagePath, &numImages);
    unsigned char *labels = LoadMnistLabels(labelPath, &numLabels);

    if (numImages != numLabels)
    {
        printf("Mismatch between images and labels count.\n");
        exit(1);
    }

    float *normalizedImg = (float *)calloc(28 * 28 * numImages, sizeof(float));
    float *onehotLabels = (float *)calloc(10 * numLabels, sizeof(float));
    // Normalize images to a 0-1 range
    Normalize(normalizedImg, images, 28 * 28 * numImages);
    // Encode labels to one ot vecotrs
    OneHotEncode(onehotLabels, labels, numLabels, 10);

    float *X = normalizedImg;
    float *y = onehotLabels;

    // Print Example Image
    printf("Example image\n");
    printf("Label[0] = %d\n", labels[0]);
    for (int i = 0; i < 28 * 28; i++)
    {
        if (normalizedImg[i] > 0)
        {
            printf("# ");
        }
        else
        {
            printf("- ");
        }
        if ((i + 1) % 28 == 0)
            printf("\n");
    }
    srand(time(NULL));

    // Build Layers
    struct Layer layerSequence[] = {
        {.inFeature = 784, .outFeature = 4096, .activation = RELU},
        {.inFeature = 4096, .outFeature = 10, .activation = NONE}};
    int numLayers = sizeof(layerSequence) / sizeof(layerSequence[0]);

    printf("Log: There are %d layers in this network\n", numLayers);

    // Intialize weight and bias values
    for (int i = 0; i < numLayers; i++)
    {
        InitWeights(&layerSequence[i]);
        InitBias(&layerSequence[i]);
    }
    int numClasses = 10;
    int totalSamples = 10000;
    int epoch_count = 20;
    int batchSize = 32;
    int batchCount = totalSamples / batchSize;

    // Allocate buffers for logits and output gradient
    float *logits = (float *)calloc(batchSize * numClasses, sizeof(float));
    float *dEdy = (float *)calloc(batchSize * numClasses, sizeof(float));

    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    for (int epoch = 0; epoch < epoch_count; epoch++)
    {
        double totalLoss = 0.0;

        for (int b = 0; b < batchCount; b++)
        {
            float *X_batch = &X[b * batchSize * 784];
            float *y_batch = &y[b * batchSize * numClasses];

            ForwardProp(logits, layerSequence, numLayers, X_batch, batchSize);
            CU_SoftMaxBatch(logits, batchSize, numClasses);
            CU_CrossEntropyGradient(dEdy, logits, y_batch, batchSize, numClasses);

            totalLoss += CalculateCCELoss(logits, y_batch, batchSize, numClasses);
            BackwardProp(layerSequence, dEdy, numLayers, batchSize, 0.05f);
        }

        printf("Epoch #%d, Avg Loss: %.6f\n", epoch + 1, totalLoss / batchCount);
    }

    clock_gettime(CLOCK_MONOTONIC, &end);

    // Calculate duration in seconds with milliseconds
    double training_time = (end.tv_sec - start.tv_sec) +
                           (end.tv_nsec - start.tv_nsec) / 1e9;

    // Print predictions and check learning result
    PredictSingle(1250, layerSequence, numLayers, X, labels, numClasses);
    PredictSingle(49000, layerSequence, numLayers, X, labels, numClasses);
    PredictSingle(2512, layerSequence, numLayers, X, labels, numClasses);

    // Free memory
    free(logits);
    free(dEdy);
    free(images);
    free(labels);
    free(normalizedImg);
    free(onehotLabels);

    printf("\nTotal training time: %.2f sec\n", training_time);
    printf("=====program end=====\n");
    return 0;
}
